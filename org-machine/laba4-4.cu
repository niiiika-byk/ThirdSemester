//технологий многопоточности для графических сопроцессоров (GPU) - CUDA
#include <iostream>
#include <string>
#include <vector>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
typedef double dtype;

__global__ void kern_dgemm(const dtype* A, const dtype* B, dtype* C, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N && j<N) {
        dtype sum = 0;
        for(int k=0;k<N;++k)
            sum += A[i*N + k] * B[k*N + j];
        C[i*N + j] = sum;
    }
}

int main(int argc, char** argv) {
    if(argc<2){ cerr<<"Usage: "<<argv[0]<<" N\n"; return 1; }
    int N = stoi(argv[1]);
    size_t sz = N * N * sizeof(dtype);

    vector<dtype> hA(N*N), hB(N*N), hC(N*N);
    mt19937_64 rng(0);
    uniform_real_distribution<dtype> dist(0.0,1.0);
    for(int i=0;i<N*N;++i){ hA[i]=dist(rng); hB[i]=dist(rng); }

    dtype *dA, *dB, *dC;
    hipMalloc(&dA, sz);
    hipMalloc(&dB, sz);
    hipMalloc(&dC, sz);
    hipMemcpy(dA, hA.data(), sz, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB.data(), sz, hipMemcpyHostToDevice);

    dim3 block(16,16);
    dim3 grid((N+15)/16, (N+15)/16);
    hipDeviceSynchronize();
    auto t0 = chrono::high_resolution_clock::now();
    kern_dgemm<<<grid, block>>>(dA, dB, dC, N);
    hipDeviceSynchronize();
    auto t1 = chrono::high_resolution_clock::now();

    hipMemcpy(hC.data(), dC, sz, hipMemcpyDeviceToHost);
    hipFree(dA); hipFree(dB); hipFree(dC);

    chrono::duration<double> dt = t1 - t0;
    cout<<"cuda, "<<N<<", "<<dt.count()<<"\n";
    return 0;
}
